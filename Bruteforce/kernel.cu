#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <fstream>
#include <iostream>
#include "../libs/src/hl_md5.h"
using namespace std;

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

//������ ������������ ��� ������� ��� ������� � ��������, �� ��� �� ���������
class __vector {
public:
    //char password[size];
    char* password;
    int* passw;
    int size;
    __device__ __host__ __vector(string pass) {
        size = pass.length();
        password = new char[size];
        strcpy(password, pass.c_str());

        passw = new int[size];
        for (int i = 0; i < size; i++)
            passw[i] = password[i];

    }

};

//���������� ������ � ��������� �� �������� � ��������� ����
thrust::host_vector<char> GetLetters() {
    vector<char> letters;
    unsigned char a;
    for (a = 65; a < 91; ++a) {
        letters.push_back(a);
    }
    for (a = 97; a < 123; ++a)
    {
        letters.push_back(a);
    }

    return letters;
}

//�������� ��� �������� ���������� �������� chars
void printCombinations(const thrust::host_vector<char>& chars, unsigned size, thrust::host_vector<char>& line, ofstream& myfile) {
    for (unsigned i = 0; i < chars.size(); i++) {
        line.push_back(chars[i]);
        if (size <= 1) { // Condition that prevents infinite loop in recursion
            for (const auto& j : line)
                myfile << j; // Simplified print to keep code shorter
            myfile << "\n";
            line.erase(line.end() - 1);
        }
        else {
            printCombinations(chars, size - 1, line, myfile); // Recursion happens here
            line.erase(line.end() - 1);
        }
    }
}
thrust::host_vector<string> FileToVector(string file_name) {

    // Open the File
    std::ifstream in(file_name);
    thrust::host_vector<string> pass_vector;
    string str;
    while (std::getline(in, str))
    {
        // Line contains string of length > 0 then save it in vector
        if (str.size() > 0)
            pass_vector.push_back(str);
    }

    return pass_vector;
}

//�����, ������� ���������� � ����� �� GPU. ������ ������� ����� ������ � ������� ������ � ��� ������(���-�� ������� = ���-�� �������) 
__global__ void SearchPassword(char **passwords, int *sizes) {
    printf("Size: %d \n", sizes[threadIdx.x]);
    //printf("Size: %c \n", passwords[0][0]);
    for(int i = 0; i < sizes[threadIdx.x]; i++)
        printf("Passw: %c \n", passwords[threadIdx.x][i]);
}

int main()
{
#pragma region MyRegion



    /*ofstream myfile;
    myfile.open("example.txt");

    vector<char> numbers = { 'A', '1', 'B', '3' };
    for (int i = 1; i < 5; i++) {
        unsigned size = i;
        thrust::host_vector<char> line;
        printCombinations(GetLetters(), size, line, myfile);
    }
    cout << endl;
    myfile.close();
    */
    /*thrust::host_vector<string> passwords = FileToVector("example.txt");
    for each (string var in passwords)
    {
        cout << var << endl;
    }*/
#pragma endregion

    //������� �������� ������� �� ����� - FileToVector(file)
    //�������� ������ � ���� - printCombinations, ����� ����������� ������� ��� �����, ��� ���� ����
  
    //thrust::host_vector<string> passwords = FileToVector("example.txt");
    //cout << passwords.size() << endl;

    //�������� ���������� ������� � ��������, ������ ������� = ������, ����� ������ ����� ���� ���������� � ����������� � ������ sizes[����� ������]
    const int lines = 10, columns = 4; 
    char** passw = new char* [lines];//������ � ��������
    int* sizes = new int[lines];//������ � ������� �����

    //������ ��������� ���������� �������, ����� ��������� ������
    for (int i = 0; i < lines; i++) {
        if (i % 2 == 0) {
            passw[i] = new char[columns];
            //col = 4;
            sizes[i] = 4;
        }
        else {
            passw[i] = new char[columns + 1];
            //col = 5;
            sizes[i] = 5;
        }
        for (int j = 0; j < sizes[i]; j++) {
            switch (j)
            {
            case 0:
                passw[i][j] = 'p';
                break;
            case 1:
                passw[i][j] = 'a';
                break;
            case 2:
                passw[i][j] = 's';
                break;
            case 3:
                passw[i][j] = 's';
                break;
            case 4:
                passw[i][j] = 'w';
                break;
            }
        }
    }
    //����� ������� � �������� ��� ��������
    /*
    for (int i = 0; i < lines; i++) {
        for (int j = 0; j < sizes[i]; j++)
        {
            cout << passw[i][j];
        }
        cout << endl;
    }
    */

    char** dev_device_passw;//��������� �� ��������� ������, ������� �� ��������� � gpu
    hipMalloc((void**)&dev_device_passw, lines * sizeof(char*));//�������� ������ �� ���������� � ��������� ��������� �� ��� � dev_device_passw. ������ �������� - ���-�� ������, �� ������ �������

    char* dev_line_passw[lines];//��������� �� ���������� ������ ���������� ��������� lines, ����� � ���� ��������� ������ ������ ��������� ������� � ��������
    for (int i = 0; i < lines; i++) {
        hipMalloc((void**)&dev_line_passw[i], sizeof(char) * sizes[i]);
        hipMemcpy(dev_line_passw[i], passw[i], sizeof(char) * sizes[i], hipMemcpyHostToDevice);
    }

    hipMemcpy(dev_device_passw, dev_line_passw, sizeof(char*) * lines, hipMemcpyHostToDevice);//�������� � ��������� dev_device_passw ��������� �� ������ dev_line_passw

    int* dev_sizes;//��������� ��� �������� ������� � ��������� �����

    hipMalloc((void**)&dev_sizes, sizeof(int) * lines);//�������� ������ �������� � ������
    hipMemcpy(dev_sizes, sizes, sizeof(int) * lines,hipMemcpyHostToDevice);//�������� ��������� �� ������ sizes 

    SearchPassword <<<1, 10 >>> (dev_device_passw, dev_sizes);//������ ������� ������� [1][10] (�.�. 1 ������ ������� �� 10 ������� = 10 �������), ������ ��� � ������ ����

    //����������� ���������� ������ �� GPU
    hipFree(dev_device_passw);
    hipFree(dev_sizes);
    hipFree(dev_line_passw);

    return 0;
}

//��� ��� ������ ���� ��� �������� �������, ����� ������������ ��� ������ �������� ���������� � GPU
__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
   
    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}


