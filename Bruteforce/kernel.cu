#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <fstream>
#include <iostream>
#include "../lib/md5.h"
#include "../libs/Bluebird Library/BB.h"
#include "../libs/cuda-hashing-algos-master/config.h"
#include "../libs/cuda-hashing-algos-master/md5.cuh"
#include "md5-cuda.cuh"
#include "md5-cuda.cu"
using namespace std;

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

//������ ������������ ��� ������� ��� ������� � ��������, �� ��� �� ���������
class __vector {
public:
    //char password[size];
    char* password;
    int* passw;
    int size;
    __device__ __host__ __vector(string pass) {
        size = pass.length();
        password = new char[size];
        strcpy(password, pass.c_str());

        passw = new int[size];
        for (int i = 0; i < size; i++)
            passw[i] = password[i];

    }

};

//���������� ������ � ��������� �� �������� � ��������� ����
thrust::host_vector<char> GetLetters() {
    vector<char> letters;
    unsigned char a;
    for (a = 65; a < 91; ++a) {
        letters.push_back(a);
    }
    for (a = 97; a < 123; ++a)
    {
        letters.push_back(a);
    }

    return letters;
}

thrust::host_vector<string> combinations;
//�������� ��� �������� ���������� �������� chars
void printCombinations(const thrust::host_vector<char>& chars, unsigned size, thrust::host_vector<char>& line) {
    for (unsigned i = 0; i < chars.size(); i++) {
        line.push_back(chars[i]);
        if (size <= 1) { // Condition that prevents infinite loop in recursion
            string pass;
            for (const auto& j : line)
                pass += j;
            //cout << pass << endl;
            combinations.push_back(pass);
            line.erase(line.end() - 1);
        }
        else {
            printCombinations(chars, size - 1, line); // Recursion happens here
            line.erase(line.end() - 1);
        }
    }
}
/*void printCombinations(const thrust::host_vector<char>& chars, unsigned size, thrust::host_vector<char>& line, ofstream& myfile) {
    for (unsigned i = 0; i < chars.size(); i++) {
        line.push_back(chars[i]);
        if (size <= 1) { // Condition that prevents infinite loop in recursion
            for (const auto& j : line)
                myfile << j; // Simplified print to keep code shorter
            myfile << "\n";
            line.erase(line.end() - 1);
        }
        else {
            printCombinations(chars, size - 1, line, myfile); // Recursion happens here
            line.erase(line.end() - 1);
        }
    }
}*/
thrust::host_vector<string> FileToVector(string file_name) {

    // Open the File
    std::ifstream in(file_name);
    thrust::host_vector<string> pass_vector;
    string str;
    while (std::getline(in, str))
    {
        // Line contains string of length > 0 then save it in vector
        if (str.size() > 0)
            pass_vector.push_back(str);
    }

    return pass_vector;
}

//�����, ������� ���������� � ����� �� GPU. ������ ������� ����� ������ � ������� ������ � ��� ������(���-�� ������� = ���-�� �������) 
__global__ void SearchPassword(char **passwords, int *sizes) {
    //printf("Size: %d \n", sizes[threadIdx.x]);
    //printf("Size: %c \n", passwords[0][0]);
   // MD5 md5;
    //cout << md5("hello");
    //printf("Size: %c \n", md5("hello"));
    for(int i = 0; i < sizes[threadIdx.x]; i++)
        printf("%c; %d \n", passwords[threadIdx.x][i], threadIdx.x);
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
int main()
{
#pragma region MyRegion

    /*ofstream myfile;
    myfile.open("example.txt");

    vector<char> numbers = { 'A', '1', 'B', '3' };
    for (int i = 1; i < 5; i++) {
        unsigned size = i;
        thrust::host_vector<char> line;
        printCombinations(GetLetters(), size, line, myfile);
    }
    cout << endl;
    myfile.close();
    */
    /*thrust::host_vector<string> passwords = FileToVector("example.txt");
    for each (string var in passwords)
    {
        cout << var << endl;
    }*/
#pragma endregion

    //������� �������� ������� �� ����� - FileToVector(file)
    //�������� ������ � ���� - printCombinations, ����� ����������� ������� ��� �����, ��� ���� ����
  
    //thrust::host_vector<string> passwords = FileToVector("example.txt");
    //cout << passwords.size() << endl;

    //�������� ���������� ������� � ��������, ������ ������� = ������, ����� ������ ����� ���� ���������� � ����������� � ������ sizes[����� ������]
    std::string data = "test";
    std::string data_hex_digest;

    md5 hash;
    hash.update(data.begin(), data.end());
    hash.hex_digest(data_hex_digest);

    std::cout << data_hex_digest << std::endl;

    MD5 md5;
    cout << md5("test");

   // mcm_cuda_md5_hash_batch(byte, word, out, outw);
    //cout << wrapper->getHashFromString("yoda");
    
    thrust::host_vector<char> pass;
    for (int i = 1; i < 3; i++) {
        unsigned size = i;
        thrust::host_vector<char> line;
        printCombinations(GetLetters(), size, line);
    }

    const int comb_lines = combinations.size();
    char** all_combinations_array = new char* [comb_lines];
    int* password_sizes = new int[comb_lines];

    for (int i = 0; i < comb_lines; i++) {
        string s = combinations[i]; 
        all_combinations_array[i] = new char[s.length()];
        strcpy(all_combinations_array[i], s.c_str());
        password_sizes[i] = s.length();
    }

    char** dev_all_combinations_array;//��������� �� ��������� ������, ������� �� ��������� � gpu
    hipMalloc((void**)&dev_all_combinations_array, comb_lines * sizeof(char*));//�������� ������ �� ���������� � ��������� ��������� �� ��� � dev_device_passw. ������ �������� - ���-�� ������, �� ������ �������


    char* dev_line[10000];//��������� �� ���������� ������ ���������� ��������� lines, ����� � ���� ��������� ������ ������ ��������� ������� � ��������
    for (int i = 0; i < comb_lines; i++) {
        hipMalloc((void**)&dev_line[i], sizeof(char) * password_sizes[i]);
        hipMemcpy(dev_line[i], all_combinations_array[i], sizeof(char) * password_sizes[i], hipMemcpyHostToDevice);
    }

    hipMemcpy(dev_all_combinations_array, dev_line, sizeof(char*) * comb_lines, hipMemcpyHostToDevice);//�������� � ��������� dev_device_passw ��������� �� ������ dev_line_passw

    int* dev_sizes;//��������� ��� �������� ������� � ��������� �����

    hipMalloc((void**)&dev_sizes, sizeof(int) * comb_lines);//�������� ������ �������� � ������
    hipMemcpy(dev_sizes, password_sizes, sizeof(int) * comb_lines, hipMemcpyHostToDevice);//�������� ��������� �� ������ sizes 

    SearchPassword << <1, 100 >> > (dev_all_combinations_array, dev_sizes);//������ ������� ������� [1][10] (�.�. 1 ������ ������� �� 10 ������� = 10 �������), ������ ��� � ������ ����

#pragma region old

    /*
    const int lines = 10, columns = 4; 
    char** passw = new char* [lines];//������ � ��������
    int* sizes = new int[lines];//������ � ������� �����

    //������ ��������� ���������� �������, ����� ��������� ������
    for (int i = 0; i < lines; i++) {
        if (i % 2 == 0) {
            passw[i] = new char[columns];
            //col = 4;
            sizes[i] = 4;
        }
        else {
            passw[i] = new char[columns + 1];
            //col = 5;
            sizes[i] = 5;
        }
        for (int j = 0; j < sizes[i]; j++) {
            switch (j)
            {
            case 0:
                passw[i][j] = 'p';
                break;
            case 1:
                passw[i][j] = 'a';
                break;
            case 2:
                passw[i][j] = 's';
                break;
            case 3:
                passw[i][j] = 's';
                break;
            case 4:
                passw[i][j] = 'w';
                break;
            }
        }
    }
    //����� ������� � �������� ��� ��������
    /*
    for (int i = 0; i < lines; i++) {
        for (int j = 0; j < sizes[i]; j++)
        {
            cout << passw[i][j];
        }
        cout << endl;
    }
    

    char** dev_device_passw;//��������� �� ��������� ������, ������� �� ��������� � gpu
    hipMalloc((void**)&dev_device_passw, lines * sizeof(char*));//�������� ������ �� ���������� � ��������� ��������� �� ��� � dev_device_passw. ������ �������� - ���-�� ������, �� ������ �������

    char* dev_line_passw[lines];//��������� �� ���������� ������ ���������� ��������� lines, ����� � ���� ��������� ������ ������ ��������� ������� � ��������
    for (int i = 0; i < lines; i++) {
        hipMalloc((void**)&dev_line_passw[i], sizeof(char) * sizes[i]);
        hipMemcpy(dev_line_passw[i], passw[i], sizeof(char) * sizes[i], hipMemcpyHostToDevice);
    }

    hipMemcpy(dev_device_passw, dev_line_passw, sizeof(char*) * lines, hipMemcpyHostToDevice);//�������� � ��������� dev_device_passw ��������� �� ������ dev_line_passw

    int* dev_sizes;//��������� ��� �������� ������� � ��������� �����

    hipMalloc((void**)&dev_sizes, sizeof(int) * lines);//�������� ������ �������� � ������
    hipMemcpy(dev_sizes, sizes, sizeof(int) * lines,hipMemcpyHostToDevice);//�������� ��������� �� ������ sizes 

    SearchPassword <<<1, 10 >>> (dev_device_passw, dev_sizes);//������ ������� ������� [1][10] (�.�. 1 ������ ������� �� 10 ������� = 10 �������), ������ ��� � ������ ����

    //����������� ���������� ������ �� GPU
    hipFree(dev_device_passw);
    hipFree(dev_sizes);
    hipFree(dev_line_passw);
    */
#pragma endregion

    return 0;
}

//��� ��� ������ ���� ��� �������� �������, ����� ������������ ��� ������ �������� ���������� � GPU
__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
   
    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}


